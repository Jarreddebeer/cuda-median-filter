
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#define BLOCKSIZE 64

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__device__ int sort(long values[], int start, int stop) {
    long pivot = values[start];
    int i = start;
    int j = stop+1;
    while (1) {
        while (values[++i] < pivot) if (i == stop) break;
        while (values[--j] >= pivot) if (j == start) break;
        if (i >= j) break;
        long tmp = values[i];
        values[i] = values[j];
        values[j] = tmp;
    }
    // place the pivot back
    long tmp = values[j];
    values[j] = pivot;
    values[start] = tmp;

    return j;
}

__device__ long getMedian(long values[], int size) {
    int start = 0;
    int stop = size - 1;
    int middle = (start + stop) / 2;
    int pivot = sort(values, start, stop);
    while (pivot != middle) {
        if (pivot > middle) {
            // median is in left half
            stop = pivot-1;
            pivot = sort(values, start, stop);
        } else {
            // median is in right half
            start = pivot+1;
            pivot = sort(values, start, stop);
        }
    }
    return values[pivot];
}

__global__ void medianFilterGPU(long* d_in, long* d_out, int histSize, int windSize) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockDim.x * blockIdx.x + tx;
    int y = blockDim.y * blockIdx.y + ty;

    // are we at a boundary?
    bool is_x_top = (tx == 0), is_x_bot = (tx == BLOCKSIZE-1);
    bool is_y_top = (ty == 0), is_y_bot = (ty == BLOCKSIZE-1);


    // window length is 9. so it overflows by 4 on each side.
    __shared__ long smem[BLOCKSIZE+8][BLOCKSIZE+8];

    // populate shared memory from histogram (histogram is padded with zeros)
    if (is_x_top) {
        for (int i = 0; i < 4; i++) smem[4+ty][i] = d_in[(y+4) * (histSize+8) + (x+4)]; // d_in[(y+4) * (histSize+8) + (x+4-(i+1))];

    } else if (is_x_bot) {
        for (int i = 0; i < 4; i++) smem[4+ty][4+BLOCKSIZE+i] = d_in[(y+4) * (histSize+8) + (x+4)];
    }
    if (is_y_top) {
        for (int i = 0; i < 4; i++) smem[i][4+tx] = d_in[(y+4) * (histSize+8) + (x+4)];

        // corner cases
        if (is_x_top) {
            for (int i = 0; i < 4; i++) {
                for (int j = 0; j < 4; j++) {
                    smem[i][j] = d_in[(y+4) * (histSize+8) + (x+4)];
                }
            }
        }

        else if (is_x_bot) {
            for (int i = 0; i < 4; i++) {
                for (int j = 0; j < 4; j++) {
                    smem[i][4+BLOCKSIZE+j] = d_in[(y+4) * (histSize+8) + (x+4)];
                }
            }
        }

    } else if (is_y_bot) {
        for (int i = 0; i < 4; i++) smem[4+BLOCKSIZE+i][4+tx] = d_in[(y+4) * (histSize+8) + (x+4)];

        // corner cases
        if (is_x_top) {
            for (int i = 0; i < 4; i++) {
                for (int j = 0; j < 4; j++) {
                    smem[4+BLOCKSIZE+i][j] = d_in[(y+4) * (histSize+8) + (x+4)];
                }
            }
        }

        else if (is_x_bot) {
            for (int i = 0; i < 4; i++) {
                for (int j = 0; j < 4; j++) {
                    smem[4+BLOCKSIZE+i][4+BLOCKSIZE+j] = d_in[(y+4) * (histSize+8) + (x+4)];
                }
            }
        }
    }

    smem[4+ty][4+tx] = d_in[(y+4) * (histSize+windSize-1) + (x+4)];

    /*
    __syncthreads();

    if (tx == 8 && ty == 8) {
        for (int i = 0; i < 24; i++) {
            for (int j = 0; j < 24; j++) {
                long val = smem[i][j];
                printf("%lu ", val);
            }
            printf("\n");
        }
    }
    */

    __syncthreads();

    // get window from shared memory
    long v[81 * sizeof(long)] = {};
    int idx = 0;
    for (int i = -4; i <= 4; i++) {
        for (int j = -4; j <= 4; j++) {
            v[idx++] = smem[(ty+4) + i][(tx+4) + j];
        }
    }

    __syncthreads();

    long med = getMedian(v, 81);
    d_out[y * histSize + x] = med;




    /*
    // first copy the data into the histograph

    hist[y * histSize + x] = histIn[y * histSize + x];

    __syncthreads();

    w_idx = 0;
    for (int dy = -windSize / 2; dy <= windSize / 2; dy++) {
        for (int dx = -windSize / 2; dx <= windSize / 2; dx++) {
            // gather the values in the window
            if (y + dy < 0) {gy = 0;}
            else if (y + dy > histSize - 1) {gy = histSize - 1;}
            else {gy = y + dy;}
            if (x + dx < 0) {gx = 0;}
            else if (x + dx > histSize - 1) {gx = histSize - 1;}
            else {gx = x + dx;}

            window[w_idx] = histIn[gy * histSize + gx];
            w_idx++;
        }
    }

    long median = getMedian(window, windSize * windSize);
    histOut[y * histSize + x] = median;

    __syncthreads();
    */

}

// read the binary file and perform binning
int readBinaryFile(const char* filename, long* grid, int histSize, int windSize) {
    printf("reading file...\n");
    int bloat = windSize / 2;
    FILE *dataFile = fopen(filename, "rb");
    if (!dataFile) {
        printf("Unable to open data file.");
        return -1;
    }
    while(!feof(dataFile)) {
        float x;
        float y;
        fread(&x, 1, sizeof(float), dataFile);
        fread(&y, 1, sizeof(float), dataFile);
        // get bins
        int xpos = (int) (x * (histSize - 1));
        int ypos = (int) (y * (histSize - 1));
        //
        grid[(ypos+bloat) * (histSize+windSize-1) + (xpos+bloat)] += 1;
    }
    fclose(dataFile);
    return 1;
}

int outputResultsToFile(const char* filename, long* grid2, int histSize) {

    double binSize = 1.0 / histSize;

    FILE *f = fopen(filename, "w");
    if (f == NULL) {
        return -1;
    }

    // print column bucket headers
    fprintf(f, ",");
    for (int x = 0; x < histSize; x++) {
        float val = binSize * x;
        if (x < histSize-1) fprintf(f, "%f,",  val);
        else                fprintf(f, "%f\n", val);
    }

    // print each row
    for (int y = 0; y < histSize; y++) {
        // first column is a bucket
        fprintf(f, "%f,", binSize * y);
        // values
        for (int x = 0; x < histSize; x++) {
            long val = grid2[y * histSize + x];
            if (x < histSize-1) fprintf(f, "%lu,",  val);
            else                fprintf(f, "%lu\n", val);
        }
    }
    fclose(f);
    return 1;
}

// read the already written CSV histogram
int readHistogramCsvFile(const char* filename, long* grid, int histSize, int windSize) {
    printf("Reading histogram file...\n");
    int bloat = windSize / 2;
    char buffer[10240];
    FILE *dataFile = fopen(filename, "r");
    if (dataFile == NULL) {
         printf("Failed to open Histogram file.");
         return -1;
    }
    char* line;
    char* value;
    int col;
    int row = 0;
    while ((line = fgets(buffer, sizeof(buffer), dataFile)) != NULL) {
        // ignore the first row, which is a header.
        if (row > 0) {
            col = 0;
            value = strtok(line, ",");
            while (value != NULL) {
                // ignore first column, which is a header
                if (col > 0) {
                    grid[((row-1)+bloat) * (histSize+windSize-1) + ((col-1)+bloat)] = atol(value);
                }
                value = strtok(NULL, ",");
                col++;
            }
        }
        row++;
    }
    return 1;
}

int main(int argc, char **argv) {

    if (argc != 3) {
        printf("Incorrect number of arguments: %d\n", argc);
        return -1;
    }

    int histSize;
    int windSize;
    sscanf(argv[1], "%d", &histSize);
    sscanf(argv[2], "%d", &windSize);

    // window size must be odd.
    if (windSize % 2 == 0) windSize++;

    // initialise the grid
    long* grid = (long*) malloc( (histSize+windSize-1)*(histSize+windSize-1) * sizeof(long));
    long* grid2 = (long*) malloc(histSize * histSize * sizeof(long));
    for (int i = 0; i < (histSize+windSize-1)*(histSize+windSize-1); i++) {
        grid[i] = 0;
    }

    // readBinaryFile("points_noise_normal.bin", grid, histSize);
    readHistogramCsvFile("gridHistogram-512.csv", grid, histSize, windSize);

    // allocate histograms to device memory
    long* d_histIn  = NULL;
    long* d_histOut = NULL;
    hipMalloc(&d_histIn,  (histSize+windSize-1) * (histSize+windSize-1) * sizeof(long));
    hipMalloc(&d_histOut, histSize * histSize * sizeof(long));

    // copy memory into device histograms
    hipMemcpy(d_histIn, grid, (histSize+windSize-1) * (histSize+windSize-1) * sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(d_histOut, grid2, histSize * histSize * sizeof(long), hipMemcpyHostToDevice);

    dim3 dimBlock = dim3(BLOCKSIZE, BLOCKSIZE, 1);
    dim3 dimGrid = dim3(
        (int) ceil(histSize / (float) dimBlock.x),
        (int) ceil(histSize / (float) dimBlock.y),
        1
    );

    printf("calling kernel...\n");
    medianFilterGPU<<<dimGrid, dimBlock>>>(d_histIn, d_histOut, histSize, windSize);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    printf("completed kernel call.\n");
    hipMemcpy(grid2, d_histOut, histSize * histSize * sizeof(long), hipMemcpyDeviceToHost);

    hipFree(d_histIn);
    hipFree(d_histOut);

    // write results to csv file
    printf("generating output...\n");
    outputResultsToFile("output.csv", grid2, histSize);
    printf("generated output\n");

    free(grid);
    free(grid2);

}
